#include "hip/hip_runtime.h"
//
#include "rtkCudaFDKWeightProjectionFilter.hcu"
#include "rtkCudaUtilities.hcu"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

texture<float, 1, hipReadModeElementType> tex_geometry; // geometry texture

inline __device__
float3 TransformIndexToPhysicalPoint(int2 idx, float3 origin, float3 row, float3 column)
{
  return make_float3(
        origin.x + row.x * idx.x + column.x * idx.y,
        origin.y + row.y * idx.x + column.y * idx.y,
        origin.z + row.z * idx.x + column.z * idx.y
        );
}

__global__
void kernel_weight_projection(
  int3 proj_size,
  float *dev_proj_in,
  float *dev_proj_out,
  float3 proj_orig,    // projection origin
  float3 proj_row,     // projection row direction & spacing
  float3 proj_col      // projection col direction & spacing
)
{
  // compute projection index (== thread index)
  int3 pIdx;
  pIdx.x = blockIdx.x * blockDim.x + threadIdx.x;
  pIdx.y = blockIdx.y * blockDim.y + threadIdx.y;
  pIdx.z = blockIdx.z * blockDim.z + threadIdx.z;
  long int pIdx_comp = pIdx.x + (pIdx.y + pIdx.z * proj_size.y)*(proj_size.x);

  // check if outside of projection grid
  if (pIdx.x >= proj_size.x || pIdx.y >= proj_size.y || pIdx.z >= proj_size.z)
    return;

  const float sdd = tex1Dfetch(tex_geometry, pIdx.z * 7 + 0);
  const float sid = tex1Dfetch(tex_geometry, pIdx.z * 7 + 1);
  const float wFac = tex1Dfetch(tex_geometry, pIdx.z * 7 + 5);
  if (sdd == 0) // parallel
  {
    dev_proj_out[pIdx_comp] = dev_proj_in[pIdx_comp] * wFac;
  }
  else // divergent
  {
    const float pOffX = tex1Dfetch(tex_geometry, pIdx.z * 7 + 2);
    const float pOffY = tex1Dfetch(tex_geometry, pIdx.z * 7 + 3);
    const float sOffY = tex1Dfetch(tex_geometry, pIdx.z * 7 + 4);
    const float tAngle = tex1Dfetch(tex_geometry, pIdx.z * 7 + 6);
    const float sina = sin(tAngle);
    const float cosa = cos(tAngle);
    const float tana = tan(tAngle);

    // compute projection point from index
    float3 pPoint = TransformIndexToPhysicalPoint(
          make_int2(pIdx.x, pIdx.y), proj_orig, proj_row, proj_col);
    pPoint.x = pPoint.x + pOffX + tana * (sdd - sid);
    pPoint.y = pPoint.y + pOffY - sOffY;

    const float numpart1 = sdd*(cosa+tana*sina);
    const float denom = sqrt((sdd * sdd + pPoint.y * pPoint.y) +
                             ((pPoint.x - sdd * tana) * (pPoint.x - sdd * tana)));
    const float cosGamma = (numpart1 - pPoint.x * sina) / denom;
    dev_proj_out[pIdx_comp] = dev_proj_in[pIdx_comp] * wFac * cosGamma;
  }
}

void
CUDA_weight_projection(
  int proj_dim[3],
  float *dev_proj_in,
  float *dev_proj_out,
  float *geometries,
  float proj_orig[3],
  float proj_row [3],
  float proj_col[3]
)
{
  // copy geometry matrix to device, bind the matrix to the texture
  float *dev_geom;
  hipMalloc((void**)&dev_geom, proj_dim[2]*7*sizeof(float));
  hipMemcpy(dev_geom, geometries, proj_dim[2]*7*sizeof(float), hipMemcpyHostToDevice);
  hipBindTexture(0, tex_geometry, dev_geom, proj_dim[2]*7*sizeof(float));

  // Thread Block Dimensions
  int tBlock_x = 16;
  int tBlock_y = 16;
  int tBlock_z = 2;

  // Each element in the volume (each voxel) gets 1 thread
  unsigned int  blocksInX = (proj_dim[0] - 1) / tBlock_x + 1;
  unsigned int  blocksInY = (proj_dim[1] - 1) / tBlock_y + 1;
  unsigned int  blocksInZ = (proj_dim[2] - 1) / tBlock_z + 1;

  dim3 dimGrid  = dim3(blocksInX, blocksInY, blocksInZ);
  dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);
  kernel_weight_projection <<< dimGrid, dimBlock >>> (
      make_int3(proj_dim[0], proj_dim[1], proj_dim[2]),
      dev_proj_in,
      dev_proj_out,
      make_float3(proj_orig[0], proj_orig[1], proj_orig[2]),
      make_float3(proj_row[0], proj_row[1], proj_row[2]),
      make_float3(proj_col[0], proj_col[1], proj_col[2])
      );

  // Unbind matrix texture
  hipUnbindTexture(tex_geometry);
  hipFree(dev_geom);
  CUDA_CHECK_ERROR;
}
