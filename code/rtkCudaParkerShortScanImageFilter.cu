#include "hip/hip_runtime.h"
//
#include "rtkCudaParkerShortScanImageFilter.hcu"
#include "rtkCudaUtilities.hcu"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

texture<float, 1, hipReadModeElementType> tex_geometry; // geometry texture

inline __device__
float3 TransformIndexToPhysicalPoint(int2 idx, float3 origin, float3 row, float3 column)
{
  return make_float3(
        origin.x + row.x * idx.x + column.x * idx.y,
        origin.y + row.y * idx.x + column.y * idx.y,
        origin.z + row.z * idx.x + column.z * idx.y
        );
}

//inline __device__
//float ToUntiltedCoordinate(float tiltedCoord, float sdd, float sx, float px, float hyp)
//{
//  return hyp * (sdd * (tiltedCoord + px) / (sdd * sdd + (sx - (tiltedCoord + px)) * sx));
//}

inline __device__
float ToUntiltedCoordinateAtIsocenter(float tiltedCoord, float sdd, float sid, float sx, float px, float sidu)
{
  // sidu is the distance between the source and the virtual untilted detector
  // l is the coordinate on the virtual detector parallel to the real detector
  // and passing at the isocenter
  const float l = (tiltedCoord + px - sx) * sid / sdd + sx;
  // a is the angle between the virtual detector and the real detector
  const float cosa = sx / sidu;
  // the following relation refers to a note by R. Clackdoyle, title
  // "Samping a tilted detector"
  return l * sid / (sidu - l * cosa);
}

__global__
void kernel_parker_weight(
  int3 proj_size,
  float *dev_proj_in,
  float *dev_proj_out,
  float delta,
  float firstAngle,
  float3 proj_orig,    // projection origin
  float3 proj_row,     // projection row direction & spacing
  float3 proj_col      // projection col direction & spacing
)
{
  // compute projection index (== thread index)
  int3 pIdx;
  pIdx.x = blockIdx.x * blockDim.x + threadIdx.x;
  pIdx.y = blockIdx.y * blockDim.y + threadIdx.y;
  pIdx.z = blockIdx.z * blockDim.z + threadIdx.z;
  long int pIdx_comp = pIdx.x + pIdx.y * proj_size.x + pIdx.z * proj_size.x * proj_size.y;

  // check if outside of projection grid
  if (pIdx.x >= proj_size.x || pIdx.y >= proj_size.y || pIdx.z >= proj_size.z)
    return;

  float sdd = tex1Dfetch(tex_geometry, pIdx.z * 5 + 0);
  float sx = tex1Dfetch(tex_geometry, pIdx.z * 5 + 1);
  float px = tex1Dfetch(tex_geometry, pIdx.z * 5 + 2);
  float sid = tex1Dfetch(tex_geometry, pIdx.z * 5 + 3);

  // convert actual index to point
  float3 pPoint = TransformIndexToPhysicalPoint(
        make_int2(pIdx.x, pIdx.y), proj_orig, proj_row, proj_col);

  // alpha projection angle
  float hyp = sqrtf(sid * sid + sx * sx); // to untilted situation
  float invsid = 1.f / hyp;
  float l = ToUntiltedCoordinateAtIsocenter(pPoint.x, sdd, sid, sx, px, hyp);
  float alpha = atan(-1 * l * invsid);

  // beta projection angle: Parker's article assumes that the scan starts at 0
  float beta = tex1Dfetch(tex_geometry, pIdx.z * 5 + 4);
  beta -= firstAngle;
  if (beta < 0)
    beta += (2.f * HIP_PI_F);

  // compute weight
  float weight = 0.;
  if (beta <= (2 * delta - 2 * alpha))
    weight = 2.f * powf(
          sinf((HIP_PI_F * beta) / (4 * (delta - alpha))),
          2.f);
  else if (beta <= (HIP_PI_F - 2 * alpha))
    weight = 2.f;
  else if (beta <= (HIP_PI_F + 2 * delta))
    weight = 2.f * powf(
          sinf((HIP_PI_F * (HIP_PI_F + 2 * delta - beta) ) / (4 * (delta + alpha))),
          2.f);

  // compute outpout by multiplying with weight
  dev_proj_out[pIdx_comp] = dev_proj_in[pIdx_comp] * weight;
}

void
CUDA_parker_weight(
  int proj_dim[3],
  float *dev_proj_in,
  float *dev_proj_out,
  float *geometries,
  float delta,
  float firstAngle,
  float proj_orig[3],
  float proj_row [3],
  float proj_col[3])
{
  // copy geometry matrix to device, bind the matrix to the texture
  float *dev_geom;
  hipMalloc((void**)&dev_geom, proj_dim[2]*5*sizeof(float));
  hipMemcpy(dev_geom, geometries, proj_dim[2]*5*sizeof(float), hipMemcpyHostToDevice);
  hipBindTexture(0, tex_geometry, dev_geom, proj_dim[2]*5*sizeof(float));

  // Thread Block Dimensions
  int tBlock_x = 16;
  int tBlock_y = 16;
  int tBlock_z = 2;

  // Each element in the volume (each voxel) gets 1 thread
  unsigned int  blocksInX = (proj_dim[0] - 1) / tBlock_x + 1;
  unsigned int  blocksInY = (proj_dim[1] - 1) / tBlock_y + 1;
  unsigned int  blocksInZ = (proj_dim[2] - 1) / tBlock_z + 1;

  dim3 dimGrid  = dim3(blocksInX, blocksInY, blocksInZ);
  dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);
  kernel_parker_weight <<< dimGrid, dimBlock >>> (
      make_int3(proj_dim[0], proj_dim[1], proj_dim[2]),
      dev_proj_in,
      dev_proj_out,
      delta, firstAngle,
      make_float3(proj_orig[0], proj_orig[1], proj_orig[2]),
      make_float3(proj_row[0], proj_row[1], proj_row[2]),
      make_float3(proj_col[0], proj_col[1], proj_col[2])
      );

  // Unbind matrix texture
  hipUnbindTexture(tex_geometry);
  hipFree(dev_geom);
  CUDA_CHECK_ERROR;
}
