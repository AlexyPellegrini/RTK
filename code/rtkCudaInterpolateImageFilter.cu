#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

// rtk includes
#include "rtkCudaInterpolateImageFilter.hcu"
#include "rtkCudaUtilities.hcu"

#include <itkMacro.h>

// cuda includes
#include <hip/hip_runtime.h>

__global__
void
weighAndAddToOutput_kernel(float *input, int4 inputSize, float* output, int phase, float weight, unsigned int Blocks_Y)
{
  // CUDA 2.0 does not allow for a 3D grid, which severely
  // limits the manipulation of large 3D arrays of data.  The
  // following code is a hack to bypass this implementation
  // limitation.
  unsigned int blockIdx_z = blockIdx.y / Blocks_Y;
  unsigned int blockIdx_y = blockIdx.y - __umul24(blockIdx_z, Blocks_Y);
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx_y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx_z, blockDim.z) + threadIdx.z;

  if (i >= inputSize.x || j >= inputSize.y || k >= inputSize.z)
      return;

  long int input_idx = ((phase * inputSize.z + k) * inputSize.y + j) * inputSize.x + i;
  long int output_idx = (k * inputSize.y + j) * inputSize.x + i;

  output[output_idx] += input[input_idx] * weight;
}

__global__
void
weighAndAddToOutput_kernel_3Dgrid(float *input, int4 inputSize, float* output, int phase, float weight)
{
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

  if (i >= inputSize.x || j >= inputSize.y || k >= inputSize.z)
      return;

  long int input_idx = ((phase * inputSize.z + k) * inputSize.y + j) * inputSize.x + i;
  long int output_idx = (k * inputSize.y + j) * inputSize.x + i;

  output[output_idx] += input[input_idx] * weight;
}



void
CUDA_interpolation(const int4 &inputSize,
                   float* input,
                   float* output,
                   int projectionNumber,
                   float **weights)
{
  // CUDA device pointers
  int    nVoxelsOutput = inputSize.x * inputSize.y * inputSize.z;
  int    memorySizeOutput = nVoxelsOutput*sizeof(float);

  // Reset output volume
  hipMemset((void *)output, 0, memorySizeOutput );

  // Thread Block Dimensions
  int tBlock_x = 16;
  int tBlock_y = 4;
  int tBlock_z = 4;
  int blocksInX = (inputSize.x - 1) / tBlock_x + 1;
  int blocksInY = (inputSize.y - 1) / tBlock_y + 1;
  int blocksInZ = (inputSize.z - 1) / tBlock_z + 1;

  int device;
  hipGetDevice(&device);

  if(CUDA_VERSION<4000 || GetCudaComputeCapability(device).first<=1)
    {
    dim3 dimGrid  = dim3(blocksInX, blocksInY*blocksInZ);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

    for (int phase=0; phase<inputSize.w; phase++)
      {
      float weight = weights[phase][projectionNumber];
      if(weight!=0)
        {
          weighAndAddToOutput_kernel <<< dimGrid, dimBlock >>> ( input,
                                                                 inputSize,
                                                                 output,
                                                                 phase,
                                                                 weight,
                                                                 blocksInY);
        }
      }
    }
  else
    {
    dim3 dimGrid  = dim3(blocksInX, blocksInY, blocksInZ);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

    for (int phase=0; phase<inputSize.w; phase++)
      {
      float weight = weights[phase][projectionNumber];
      if(weight!=0)
        {
          weighAndAddToOutput_kernel_3Dgrid <<< dimGrid, dimBlock >>> ( input,
                                                                       inputSize,
                                                                       output,
                                                                       phase,
                                                                       weight);
        }
      }
    }

}
