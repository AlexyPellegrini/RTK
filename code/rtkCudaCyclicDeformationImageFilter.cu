#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

// rtk includes
#include "rtkCudaCyclicDeformationImageFilter.hcu"
#include "rtkCudaUtilities.hcu"

#include <itkMacro.h>

// cuda includes
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// TEXTURES AND CONSTANTS //

__constant__ int4 c_inputSize;

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

void
CUDA_linear_interpolate_along_fourth_dimension(unsigned int inputSize[4],
                                              float* input,
                                              float* output,
                                              unsigned int frameInf,
                                              unsigned int frameSup,
                                              double weightInf,
                                              double weightSup)
{
  hipblasHandle_t  handle;
  hipblasCreate(&handle);

  float wInf = (float) weightInf;
  float wSup = (float) weightSup;

  int numel = inputSize[0] * inputSize[1] * inputSize[2];

  hipMemset((void *)output, 0, numel * sizeof(float) * 3);

  for (unsigned int component = 0; component < 3; component++)
    {
    // Create a pointer to the component-th component of frameInf in the input,
    // and add it weightInf times to the component-th component of the output
    float * pinf = input + frameInf * numel * 3 + component;
    hipblasSaxpy(handle, numel, &wInf, pinf, 3, output + component, 3);

    // Create a pointer to the component-th component of frameSup in the input,
    // and add it weightSup times to the component-th component of the output
    float * psup = input + frameSup * numel * 3 + component;
    hipblasSaxpy(handle, numel, &wSup, psup, 3, output + component, 3);
    }

  // Destroy Cublas context
  hipblasDestroy(handle);

  CUDA_CHECK_ERROR;
}
