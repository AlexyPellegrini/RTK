#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
#include "rtkConfiguration.h"

/****************************************************\
* Uncomment the line below to enable verbose output. *
* Enabling this should not nerf performance.         *
\****************************************************/
//#define VERBOSE

/**********************************************************\
* Uncomment the line below to enable detailed performance  *
* reporting.  This measurement alters the system, however, *
* resulting in significantly slower kernel execution.      *
\**********************************************************/
//#define TIME_KERNEL

/*****************
*  C   #includes *
*****************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

/*****************
* CUDA #includes *
*****************/
#include <hip/hip_runtime.h>

/*****************
* FDK  #includes *
*****************/
#include "cuda_util.h"
#include "itkCudaFDKBackProjectionImageFilter.hcu"

// P R O T O T Y P E S ////////////////////////////////////////////////////
__global__ void kernel_fdk (float *dev_vol, int2 img_dim, int3 vol_dim, unsigned int Blocks_Y, float invBlocks_Y);
///////////////////////////////////////////////////////////////////////////

// T E X T U R E S ////////////////////////////////////////////////////////
texture<float, 1, hipReadModeElementType> tex_img;
texture<float, 1, hipReadModeElementType> tex_matrix;
///////////////////////////////////////////////////////////////////////////

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

__global__
void kernel_fdk_gmem (
    float *dev_vol,
    float *pimg,
    float *pmat,
    int2 img_dim,
    int3 vol_dim,
    unsigned int Blocks_Y,
    float invBlocks_Y)
{
  // CUDA 2.0 does not allow for a 3D grid, which severely
  // limits the manipulation of large 3D arrays of data.  The
  // following code is a hack to bypass this implementation
  // limitation.
  unsigned int blockIdx_z = __float2uint_rd(blockIdx.y * invBlocks_Y);
  unsigned int blockIdx_y = blockIdx.y - __umul24(blockIdx_z, Blocks_Y);
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx_y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx_z, blockDim.z) + threadIdx.z;

  if (i >= vol_dim.x || j >= vol_dim.y || k >= vol_dim.z) {
      return; 
  }

  // Index row major into the volume
  long int vol_idx = i + ( j*(vol_dim.x) ) + ( k*(vol_dim.x)*(vol_dim.y) );

  float3 ip;
  int2 ip_r;
  float voxel_data;

  // matrix multiply
  ip.x = pmat[0]*i + pmat[1]*j + pmat[2]*k + pmat[3];
  ip.y = pmat[4]*i + pmat[5]*j + pmat[6]*k + pmat[7];
  ip.z = pmat[8]*i + pmat[9]*j + pmat[10]*k + pmat[11];

  // Change coordinate systems
  ip.z = 1 / ip.z;
  ip.x = ip.x * ip.z;
  ip.y = ip.y * ip.z;

  // Get pixel from 2D image
  ip_r.x = __float2int_rd(ip.x);
  ip_r.y = __float2int_rd(ip.y);

  // Clip against image dimensions
  if (ip_r.x < 0 || ip_r.x >= img_dim.x || ip_r.y < 0 || ip_r.y >= img_dim.y) {
      return;
  }
  voxel_data = pimg[ip_r.x*img_dim.x + ip_r.y];

  // Place it into the volume
  dev_vol[vol_idx] += ip.z * ip.z * voxel_data;
}


__global__
void kernel_fdk (float *dev_vol, int2 img_dim, int3 vol_dim, unsigned int Blocks_Y, float invBlocks_Y)
{
  // CUDA 2.0 does not allow for a 3D grid, which severely
  // limits the manipulation of large 3D arrays of data.  The
  // following code is a hack to bypass this implementation
  // limitation.
  unsigned int blockIdx_z = __float2uint_rd(blockIdx.y * invBlocks_Y);
  unsigned int blockIdx_y = blockIdx.y - __umul24(blockIdx_z, Blocks_Y);
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx_y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx_z, blockDim.z) + threadIdx.z;

  if (i >= vol_dim.x || j >= vol_dim.y || k >= vol_dim.z) {
      return; 
  }

  // Index row major into the volume
  long int vol_idx = i + ( j*(vol_dim.x) ) + ( k*(vol_dim.x)*(vol_dim.y) );

  float3 ip;
  float voxel_data;

  // matrix multiply
  ip.x = tex1Dfetch(tex_matrix, 0)*i + tex1Dfetch(tex_matrix, 1)*j + tex1Dfetch(tex_matrix, 2)*k + tex1Dfetch(tex_matrix, 3);
  ip.y = tex1Dfetch(tex_matrix, 4)*i + tex1Dfetch(tex_matrix, 5)*j + tex1Dfetch(tex_matrix, 6)*k + tex1Dfetch(tex_matrix, 7);
  ip.z = tex1Dfetch(tex_matrix, 8)*i + tex1Dfetch(tex_matrix, 9)*j + tex1Dfetch(tex_matrix, 10)*k + tex1Dfetch(tex_matrix, 11);

  // Change coordinate systems
  ip.z = 1 / ip.z;
  ip.x = ip.x * ip.z;
  ip.y = ip.y * ip.z;

  // Get pixel from 2D image
  ip.x = __float2int_rd(ip.x);
  ip.y = __float2int_rd(ip.y);

  // Clip against image dimensions
  if (ip.x < 0 || ip.x >= img_dim.x || ip.y < 0 || ip.y >= img_dim.y) {
      return;
  }
  voxel_data = tex1Dfetch(tex_img, ip.y*img_dim.x + ip.x);

  // Place it into the volume
  dev_vol[vol_idx] += ip.z * ip.z * voxel_data;
}
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_reconstruct_conebeam_init() /////////////////////////////
extern "C"
int
CUDA_reconstruct_conebeam_init (
  kernel_args_fdk *kargs,
  kernel_args_fdk *&dev_kargs, // Holds kernel parameters on device
  float *&dev_vol,             // Holds voxels on device
  float *&dev_img,             // Holds image pixels on device
  float *&dev_matrix           // Holds matrix on device
)
{
  // Size of volume Malloc
  int vol_size_malloc = (kargs->vol_dim.x*kargs->vol_dim.y*kargs->vol_dim.z)*sizeof(float);

  // CUDA device pointers
  hipMalloc( (void**)&dev_matrix, 12*sizeof(float) );
  hipMalloc( (void**)&dev_kargs, sizeof(kernel_args_fdk) );

/*
  ////// TIMING CODE //////////////////////
  // Initialize Windows HighRes Timer
  Timer timer_total;
  double time_total = 0;
  #if defined (TIME_KERNEL)
  Timer timer;
  double backproject_time = 0.0;
  double filter_time = 0.0;
  double io_time = 0.0;
  #endif

  // Start timing total execution
  plm_timer_start (&timer_total);
*/

#if defined (VERBOSE)
  // First, we need to allocate memory on the host device
  // for the 3D volume of voxels that will hold our reconstruction.
  printf("========================================\n");
  printf("Allocating %dMB of video memory...", vol_size_malloc/1048576);
#endif

  hipMalloc( (void**)&dev_vol, vol_size_malloc);
  hipMemset( (void *) dev_vol, 0, vol_size_malloc);  
  CUDA_check_error("Unable to allocate data volume");

#if defined (VERBOSE)
  printf(" done.\n\n");

  // State the kernel execution parameters
  printf("kernel parameters:\n dimGrid: %u, %u "
  "(Logical: %u, %u, %u)\n dimBlock: %u, %u, %u\n", 
  dimGrid.x, dimGrid.y, dimGrid.x, blocksInY, blocksInZ, 
  dimBlock.x, dimBlock.y, dimBlock.z);
  printf("%u voxels in volume\n", vol->npix);
  printf("%u projections to process\n", 1+(options->last_img - options->first_img) / options->skip_img);
  printf("%u Total Operations\n", vol->npix * (1+(options->last_img - options->first_img) / options->skip_img));
  printf("========================================\n\n");

  // Start working
  printf("Processing...\n");
#endif

  // This is just to retrieve the 2D image dimensions
  hipMalloc ((void**)&dev_img, kargs->img_dim.x*kargs->img_dim.y*sizeof(float));

  return 0;
}



///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_reconstruct_conebeam() //////////////////////////////////
extern "C"
int 
CUDA_reconstruct_conebeam (
    float *vol,
    float *proj,
    kernel_args_fdk *kargs,
    kernel_args_fdk *dev_kargs,
    float *dev_vol,
    float *dev_img,
    float *dev_matrix
)
{
  // Size of volume Malloc
  int vol_size_malloc = (kargs->vol_dim.x*kargs->vol_dim.y*kargs->vol_dim.z)*sizeof(float);

  // Project each image into the volume one at a time
  //for (int i = 0; i < proj_dir->num_proj_images; i++) {

  // Thread Block Dimensions
  int tBlock_x = 16;
  int tBlock_y = 4;
  int tBlock_z = 4;

  // Each element in the volume (each voxel) gets 1 thread
  int blocksInX = (kargs->vol_dim.x+tBlock_x-1)/tBlock_x;
  int blocksInY = (kargs->vol_dim.y+tBlock_y-1)/tBlock_y;
  int blocksInZ = (kargs->vol_dim.z+tBlock_z-1)/tBlock_z;
  dim3 dimGrid  = dim3(blocksInX, blocksInY*blocksInZ);
  dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

  // Copy image pixel data & projection matrix to device Global Memory
  // and then bind them to the texture hardware.
  hipMemcpy (dev_img, proj, kargs->img_dim.x*kargs->img_dim.y*sizeof(float), hipMemcpyHostToDevice);
  hipBindTexture (0, tex_img, dev_img, kargs->img_dim.x*kargs->img_dim.y*sizeof(float));
  hipMemcpy (dev_matrix, kargs->matrix, sizeof(kargs->matrix), hipMemcpyHostToDevice);
  hipBindTexture (0, tex_matrix, dev_matrix, sizeof(kargs->matrix));

  #if defined (VERBOSE)
  printf ("Executing kernel... ");
  #endif
/*
  #if defined (TIME_KERNEL)
  plm_timer_start (&timer);
  #endif
*/
  // Note: cbi->img AND cbi->matrix are passed via texture memory
  //-------------------------------------
  kernel_fdk <<< dimGrid, dimBlock >>> (
      dev_vol,
      kargs->img_dim,
      kargs->vol_dim,
      blocksInY,
      1.0f/(float)blocksInY
  );

  CUDA_check_error("Kernel Panic!");

  #if defined (TIME_KERNEL)
  // CUDA kernel calls are asynchronous...
  // In order to accurately time the kernel
  // execution time we need to set a thread
  // barrier here after its execution.
  hipDeviceSynchronize();
  #endif

  #if defined (VERBOSE)
  printf ("done.\n");
  #endif

  // Unbind the image and projection matrix textures
  hipUnbindTexture (tex_img);
  hipUnbindTexture (tex_matrix);
/*
  #if defined (TIME_KERNEL)
  backproject_time += plm_timer_report (&timer);
  #endif
*/

  #if defined (VERBOSE)
  printf(" done.\n\n");
  #endif
  
  // Copy reconstructed volume from device to host
  hipMemcpy (vol, dev_vol, vol_size_malloc, hipMemcpyDeviceToHost);
  CUDA_check_error ("Error: Unable to retrieve data volume.");

/*    
  // Report total time
  time_total = plm_timer_report (&timer_total);
  printf ("========================================\n");
  printf ("[Total Execution Time: %.9fs ]\n", time_total);
  #if defined (TIME_KERNEL)
  printf ("I/O time = %g\n", io_time);
  printf ("Filter time = %g\n", filter_time);
  printf ("Backprojection time = %g\n", backproject_time);
  #endif
*/
/*
  int num_images = 1 + (options->last_img - options->first_img) 
  / options->skip_img;
  printf ("[Average Projection Time: %.9fs ]\n", time_total / num_images);
  #if defined (TIME_KERNEL)
  printf ("I/O time = %g\n", io_time / num_images);
  printf ("Filter time = %g\n", filter_time / num_images);
  printf ("Backprojection time = %g\n", backproject_time / num_images);
  #endif
  printf ("========================================\n");
*/
  return 0;
}

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_reconstruct_conebeam_cleanup() //////////////////////////
int
CUDA_reconstruct_conebeam_cleanup (
  kernel_args_fdk *dev_kargs,
  float *dev_vol,
  float *dev_img,
  float *dev_matrix
)
{
  // Cleanup
  hipFree (dev_img);
  hipFree (dev_kargs);
  hipFree (dev_matrix);
  hipFree (dev_vol); 

  return 0;
}
