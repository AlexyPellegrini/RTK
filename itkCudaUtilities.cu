#include "itkCudaUtilities.hcu"
#include <itkMacro.h>

void
CUDA_check_error (const std::string &msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
      itkGenericExceptionMacro(<< "CUDA ERROR: " << msg << " (" << err << ")." << std::endl);
}
