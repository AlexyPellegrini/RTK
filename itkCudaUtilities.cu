#include "itkCudaUtilities.hcu"

std::vector<int> GetListOfCudaDevices()
{
  std::vector<int>      deviceList;
  int                   deviceCount;
  struct hipDeviceProp_t properties;
  hipError_t           cudaResultCode = hipGetDeviceCount(&deviceCount);
  if (cudaResultCode == hipSuccess)
    {
    for (int device = 0; device < deviceCount; ++device) {
      hipGetDeviceProperties(&properties, device);
      if (properties.major != 9999)   /* 9999 means emulation only */
        deviceList.push_back(device);
      }
    }
  if(deviceList.size()<1)
    itkGenericExceptionMacro(<< "No CUDA device available");

  return deviceList;
}
